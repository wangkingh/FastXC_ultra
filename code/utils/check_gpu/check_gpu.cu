#include<hip/hip_runtime.h>
#include<stdio.h>

int main(){
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int n=0;n<nDevices;n++){
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop,n);
		printf("Device Number: %d\n",n);
		printf(" Device  name: %s\n",prop.name);
		printf(" Compute capability:%d.%d\n",prop.major,prop.minor);
	}
	return 0;
}
