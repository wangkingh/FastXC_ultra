#include "hip/hip_runtime.h"
/* last updated by wangjx@20250421 */

#include "cuda.processing.cuh"
#include "cuda.util.cuh"

extern "C"
{
#include "arguproc.h"
#include "design_filter_response.h"
#include "in_out_node.h"
#include "cal_nseg.h"
#include "read_filelist.h"
#include "par_rw_data.h"
#include "sac.h"
#include "util.h"
}

int main(int argc, char **argv)
{
    // Parsing arguments
    ARGUTYPE argument;
    ArgumentProcess(argc, argv, &argument);
    PathNode *pInFileList = readPathList(argument.sac_lst);   // Read in sac path list file
    PathNode *pOutFileList = readPathList(argument.spec_lst); // Read in spec output path list file

    createDirectories(pOutFileList); // Create output_dir for .segspec files

    FilePathArray InPaths = PathList2Array(pInFileList); // Turn file chain input list chain into array
    FilePathArray OutPaths = PathList2Array(pOutFileList);

    int num_ch = argument.num_ch; // Get the number of channels

    // Parsing Whiten and Normalization Type
    int wh_before = 0, wh_after = 0, do_runabs_mf = 0, do_runabs = 0, do_onebit = 0;
    switch (argument.whitenType)
    {
    case 0:
        wh_before = 0, wh_after = 0;
        break;
    case 1:
        wh_before = 1;
        break;
    case 2:
        wh_after = 1;
        break;
    case 3:
        wh_before = 1, wh_after = 1;
        break;
    default:
        printf("Invalid value for bandwhiten\n");
    }

    switch (argument.normalizeType)
    {
    case 0:
        do_runabs_mf = 0, do_onebit = 0, do_runabs = 0;
        break;
    case 1:
        do_runabs_mf = 1;
        break;
    case 2:
        do_onebit = 1;
        break;
    case 3:
        do_runabs = 1;
        break;
    default:
        printf("Invalid value for normalization\n");
    }

    size_t nValid_sacnum = InPaths.count;         // Count the number of input files
    size_t nValid_batch = nValid_sacnum / num_ch; // Count the number of input files/channels

    size_t gpu_id = argument.gpu_id; // Set the gpu_id
    hipSetDevice(gpu_id);

    SACHEAD sachd;
    if (read_sachead(pInFileList->path, &sachd) != 0)
    {
        fprintf(stderr, "ERROR reading first SACHEAD\n");
        exit(EXIT_FAILURE);
    }
    int npts = sachd.npts; // Read npts and delta form the file file of pInFileList
    float delta = sachd.delta;

    int segment_pts_1x = cal_segment_length(argument.seglen, npts, delta); // Calculate the number of points of sement

    int shift_length_pts = (int)(argument.segshift / delta + 0.5f);
    if (shift_length_pts < 1)
        shift_length_pts = segment_pts_1x;
    int nstep = 0;
    if (segment_pts_1x <= npts)
    {
        // (npts - window_length_pts) / shift_length_pts + 1
        nstep = (npts - segment_pts_1x) / shift_length_pts + 1;
        if (nstep < 1)
            nstep = 1;
    }
    else
    {
        nstep = 1; // if window_length_pts > npts, only one segment
    }
    printf("nstep = %d\n", nstep);

    int segment_pts_2x = segment_pts_1x * 2; // Create segment_pts_2x for zero padding data
    int nspec_output = segment_pts_2x / 2 + 1;

    float df_1x = 1.0 / (segment_pts_1x * delta);
    float df_2x = 1.0 / (segment_pts_2x * delta);

    // Parse frequcny band, calculate idx of corner and cutoff frequency
    float freq_low = argument.freq_low;
    float freq_high = argument.freq_high;
    int f_idx1 = int(freq_low * 0.667 / df_1x);
    int f_idx2 = int(freq_low / df_1x);
    int f_idx3 = int(freq_high / df_1x);
    int f_idx4 = int(freq_high * 1.333 / df_1x);

    int filter_count = 0;
    ButterworthFilter *filter = readButterworthFilters(argument.filter_file, &filter_count);     // read in filter file
    FilterResp *myResp = processButterworthFilters(filter, filter_count, df_2x, segment_pts_2x); // Calculate filter f domain response

    // Parsing skip_steps
    int *skip_steps = argument.skip_steps;
    int skip_step_count = argument.skip_step_count;
    int nstep_valid = nstep;

    int skip_flags[nstep]; // Mark which steps should be skipped
    memset(skip_flags, false, sizeof(skip_flags));

    for (int i = 0; i < skip_step_count; ++i)
    {
        int skip_step = skip_steps[i];
        if (skip_step >= 0 && skip_step < nstep)
        {
            skip_flags[skip_step] = true;
            printf("Step [no. %d] will be skipped \n", skip_step);
        }
    }

    nstep_valid = 0; // Calculate validated steps
    for (int i = 0; i < nstep; ++i)
    {
        if (!skip_flags[i])
        {
            ++nstep_valid;
        }
    }

    // ********* Calculate CPU memory *********** //
    float *h_sacdata = NULL;
    complex *h_spectrum = NULL;

    InOutNode *pInOutList = NULL;

    size_t unit_sacdata_size = npts * sizeof(float);                          // input sac data
    size_t unit_spectrum_size = nstep_valid * nspec_output * sizeof(complex); // output total spectrum
    size_t unit_InOutNode_size = sizeof(InOutNode);                           // contain head/path/data
    size_t unit_thread_write_size = sizeof(thread_info_write);                // contain thread_info_write
    size_t unit_thread_read_size = sizeof(thread_info_read);                  // contain thread_info_read
    // print unit size of each part in MB
    printf("unit_sacdata_size = %ld MB\n", unit_sacdata_size / 1024 / 1024);
    printf("unit_spectrum_size = %ld MB\n", unit_spectrum_size / 1024 / 1024);
    printf("unit_InOutNode_size = %ld MB\n", unit_InOutNode_size / 1024 / 1024);
    printf("unit_thread_write_size = %ld MB\n", unit_thread_write_size / 1024 / 1024);

    size_t unitCpuRam = num_ch * (unit_sacdata_size +
                                  unit_spectrum_size +
                                  unit_InOutNode_size +
                                  unit_thread_write_size +
                                  unit_thread_read_size);

    size_t h_batch = EstimateCpuBatch(unitCpuRam, argument.gpu_num);
    printf("CPU: h_batch = %ld\n", h_batch);
    // Allocate memory for GPU
    size_t wh_flag = wh_after || wh_before;
    size_t d_batch = EstimateGpuBatch(gpu_id, npts, segment_pts_1x, nstep_valid, num_ch, filter_count, wh_flag, do_runabs_mf || do_runabs);
    // print vallid batch
    printf("GPU no.%zu: nValid_batch = %ld\n", gpu_id, nValid_batch);
    h_batch = (h_batch > nValid_batch) ? nValid_batch : h_batch;
    printf("GPU no.%zu: d_batch = %ld, h_batch = %ld, num_ch = %d\n", gpu_id, d_batch, h_batch, num_ch);
    d_batch = (d_batch > h_batch) ? h_batch : d_batch;
    h_batch = d_batch;
    size_t batch = h_batch;
    batch = batch < 1 ? 1 : batch;

    printf("[GPU no.%zu]: batch = %ld, num_ch=%d\n", gpu_id, batch, num_ch);

    // Allocate CPU memory
    CpuMalloc((void **)&pInOutList, num_ch * batch * unit_InOutNode_size);
    CpuMalloc((void **)&h_sacdata, num_ch * batch * unit_sacdata_size);
    CpuMalloc((void **)&h_spectrum, num_ch * batch * unit_spectrum_size);

    // Initializing the memory for InOutNode, batch *num_ch datas will be processed at same time
    size_t sacpathSize = MAXPATH * sizeof(char);
    size_t specpathSize = MAXPATH * sizeof(char);
    size_t spechdSize = sizeof(SEGSPEC);
    size_t sachdSize = sizeof(SACHEAD);
    for (size_t i = 0; i < batch * num_ch; i++)
    {
        CpuMalloc((void **)&(pInOutList[i].sacpath), sacpathSize);
        CpuMalloc((void **)&(pInOutList[i].specpath), specpathSize);

        CpuMalloc((void **)&(pInOutList[i].sac_hd), sachdSize);
        CpuMalloc((void **)&(pInOutList[i].segspec_hd), spechdSize);

        pInOutList[i].sac_data = h_sacdata + i * npts;
        pInOutList[i].spectrum = h_spectrum + i * nstep_valid * nspec_output;

        pInOutList[i].nspec = nspec_output;
        pInOutList[i].nstep = nstep_valid;
        pInOutList[i].df = df_2x;
        pInOutList[i].dt = delta;
    }

    // ********** Allocate memory for GPU ************
    float *d_sacdata = NULL;          // segment sacdata in GPU
    float *d_sacdata_2x = NULL;       // 2x  length segment sacdata in GPU
    float *d_filtered_sacdata = NULL; // filteredsegment sacdata in GPU
    float *d_total_sacdata = NULL;    // summed sacdata of all bands in GPU

    hipComplex *d_spectrum = NULL;    // segment spectrum in GPU
    hipComplex *d_spectrum_2x = NULL; // 2x  length segment spectrum in GPU

    float *d_weight = NULL;     // weight of each segment spectrum data and sac data
    float *d_tmp = NULL;        // used in runabs
    float *d_tmp_weight = NULL; // used in runabs and whiten, store weight of single channel data

    hipComplex *d_responses = NULL; // butterworth 2th filter responses, zero-padded segment length

    double *d_sum = NULL;  // used in rtr and rdc
    double *d_isum = NULL; // used in rtr and rdc

    hipfftHandle planfwd;    // forward fft
    hipfftHandle planinv;    // inverse fft
    hipfftHandle planfwd_2x; // forward fft, for 2x zero-padding series
    hipfftHandle planinv_2x; // inverse fft, for 2x zero-padding series

    // Allocating memory in GPU Device
    AllocateGpuMemory(batch, segment_pts_1x, num_ch, do_runabs || do_runabs_mf, wh_flag,
                      &d_sacdata, &d_spectrum,
                      &d_sacdata_2x, &d_spectrum_2x,
                      &d_filtered_sacdata,
                      &d_total_sacdata,
                      &d_responses, &d_tmp,
                      &d_weight, &d_tmp_weight,
                      filter_count, &d_sum, &d_isum,
                      &planfwd, &planinv,
                      &planfwd_2x, &planinv_2x);

    // copy each filter response to GPU and store freq_low array
    float freq_lows[filter_count];
    for (int i = 0; i < filter_count; i++)
    {
        freq_lows[i] = myResp[i].freq_low;
        CUDACHECK(hipMemcpy(d_responses + i * segment_pts_2x, myResp[i].response, segment_pts_2x * sizeof(hipComplex), hipMemcpyHostToDevice));
    }

    // ********** MAIN PROCESSING PART ********** /
    for (size_t finish_batch = 0; finish_batch < nValid_batch; finish_batch += batch)
    {
        size_t proc_batch = (finish_batch + batch > nValid_batch) ? nValid_batch - finish_batch : batch;

        // Setup Input Output Chain
        size_t finish_cnt = finish_batch * num_ch;
        size_t proc_cnt = proc_batch * num_ch;
        for (size_t i = finish_cnt, j = 0; i < finish_cnt + proc_cnt; i++, j++)
        {
            pInOutList[j].sacpath = InPaths.paths[i];
            pInOutList[j].specpath = OutPaths.paths[i];
        }
        memset(h_sacdata, 0, proc_cnt * npts * sizeof(float));
        memset(h_spectrum, 0, proc_cnt * nstep_valid * nspec_output * sizeof(complex));

        ThreadPoolRead *read_pool = create_threadpool_read(argument.thread_num);
        ThreadPoolWrite *write_pool = create_threadpool_write(argument.thread_num);
        parallel_read_sac(read_pool, proc_cnt, pInOutList, argument.thread_num); // Parallel read SAC data

        dim3 dimgrd_1x, dimblk_1x, dimgrd_2x, dimblk_2x;
        DimCompute(&dimgrd_1x, &dimblk_1x, segment_pts_1x, proc_cnt);
        DimCompute(&dimgrd_2x, &dimblk_2x, segment_pts_2x, proc_cnt);
        for (int stepidx = 0, done_step = 0; stepidx < nstep; stepidx++)
        {
            int skip_this_step = 0;
            for (int flag_i = 0; flag_i < argument.skip_step_count; flag_i++)
            {
                if (argument.skip_steps[flag_i] == stepidx)
                {
                    skip_this_step = 1;
                    break;
                }
            }

            if (skip_this_step)
            {
                continue;
            }

            // clean d_sacdata_* and d_spectrum_*
            CUDACHECK(hipMemset(d_sacdata, 0, proc_cnt * segment_pts_1x * sizeof(float)));
            CUDACHECK(hipMemset(d_spectrum, 0, proc_cnt * segment_pts_1x * sizeof(hipComplex)));
            CUDACHECK(hipMemset(d_sacdata_2x, 0, proc_cnt * segment_pts_2x * sizeof(float)));
            CUDACHECK(hipMemset(d_spectrum_2x, 0, proc_cnt * segment_pts_2x * sizeof(hipComplex)));

            // copy sacdata to GPU
            CUDACHECK(hipMemcpy2D(d_sacdata, segment_pts_1x * sizeof(float),
                                   h_sacdata + stepidx * shift_length_pts, npts * sizeof(float),
                                   segment_pts_1x * sizeof(float), proc_cnt, hipMemcpyHostToDevice));
            preprocess(d_sacdata, d_sum, d_isum, segment_pts_1x, proc_cnt, freq_low, delta); // isnan,rtr,reman,taper

            // double zero-padding before filtering, add 20241216 by wangjx
            CUDACHECK(hipMemcpy2D(d_sacdata_2x, segment_pts_2x * sizeof(float), d_sacdata, segment_pts_1x * sizeof(float), segment_pts_1x * sizeof(float), proc_cnt, hipMemcpyDeviceToDevice));
            CUFFTCHECK(hipfftExecR2C(planfwd_2x, (hipfftReal *)d_sacdata_2x, (hipfftComplex *)d_spectrum_2x));
            FwdNormalize2DKernel<<<dimgrd_2x, dimblk_2x>>>(d_spectrum_2x, segment_pts_2x, segment_pts_2x, proc_cnt, delta);
            cisnan2DKernel<<<dimgrd_2x, dimblk_2x>>>(d_spectrum_2x, segment_pts_2x, segment_pts_2x, proc_cnt);
            dim3 c_fdimgrd, c_fdimblk;                   // Filtering add 20240711 by wangjx
            size_t fwidth_2x = 0.5 * segment_pts_2x + 1; // set to 0.5 * segment_pts_2x + 1, because of the zero-padding
            DimCompute(&c_fdimgrd, &c_fdimblk, fwidth_2x, proc_cnt);
            filterKernel<<<c_fdimgrd, c_fdimblk>>>(d_spectrum_2x, d_responses, segment_pts_2x, fwidth_2x, proc_cnt);
            CUFFTCHECK(hipfftExecC2R(planinv_2x, (hipfftComplex *)d_spectrum_2x, (hipfftReal *)d_sacdata_2x));
            InvNormalize2DKernel<<<dimgrd_2x, dimblk_2x>>>(d_sacdata_2x, segment_pts_2x, segment_pts_2x, proc_cnt, delta);

            // cut the data to nseg length
            CUDACHECK(hipMemcpy2D(d_sacdata, segment_pts_1x * sizeof(float), d_sacdata_2x, segment_pts_2x * sizeof(float), segment_pts_1x * sizeof(float), proc_cnt, hipMemcpyDeviceToDevice));

            if (wh_before)
            {
                CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_sacdata, (hipfftComplex *)d_spectrum));
                FwdNormalize2DKernel<<<dimgrd_1x, dimblk_1x>>>(d_spectrum, segment_pts_1x, segment_pts_1x, proc_cnt, delta);
                freqWhiten(d_spectrum, d_weight, d_tmp_weight, d_tmp, num_ch, segment_pts_1x, proc_batch, delta, f_idx1, f_idx2, f_idx3, f_idx4);
                cisnan2DKernel<<<dimgrd_1x, dimblk_1x>>>(d_spectrum, segment_pts_1x, segment_pts_1x, proc_cnt);
                CUFFTCHECK(hipfftExecC2R(planinv, (hipfftComplex *)d_spectrum, (hipfftReal *)d_sacdata));
                InvNormalize2DKernel<<<dimgrd_1x, dimblk_1x>>>(d_sacdata, segment_pts_1x, segment_pts_1x, proc_cnt, delta);
            }

            //   Time Domain Normalization
            if (do_runabs_mf)
            {
                runabs_mf(d_sacdata, d_filtered_sacdata, d_total_sacdata,
                          d_sacdata_2x, d_spectrum_2x,
                          d_responses, d_tmp,
                          d_weight, d_tmp_weight,
                          &planinv, freq_lows,
                          filter_count, delta, proc_batch, num_ch, MAXVAL, segment_pts_1x, segment_pts_2x, &planinv_2x, &planfwd_2x);
                CUDACHECK(hipMemcpy2D(d_sacdata, segment_pts_1x * sizeof(float),
                                       d_total_sacdata, segment_pts_1x * sizeof(float),
                                       segment_pts_1x * sizeof(float), proc_cnt, hipMemcpyDeviceToDevice));
            }

            if (do_onebit)
            {
                onebit2DKernel<<<dimgrd_1x, dimblk_1x>>>(d_sacdata, segment_pts_1x, segment_pts_1x, proc_cnt);
            }

            if (do_runabs)
            {
                float freq_lows_limit = freq_low * 0.667;
                runabs(d_sacdata, d_tmp, d_weight, d_tmp_weight, freq_lows_limit, delta, proc_batch, num_ch, segment_pts_1x, MAXVAL);
            }

            if (wh_after)
            {
                CUFFTCHECK(hipfftExecR2C(planfwd, (hipfftReal *)d_sacdata, (hipfftComplex *)d_spectrum));
                FwdNormalize2DKernel<<<dimgrd_1x, dimblk_1x>>>(d_spectrum, segment_pts_1x, segment_pts_1x, proc_cnt, delta);
                freqWhiten(d_spectrum, d_weight, d_tmp_weight, d_tmp, num_ch, segment_pts_1x, proc_batch, delta, f_idx1, f_idx2, f_idx3, f_idx4);
                cisnan2DKernel<<<dimgrd_1x, dimblk_1x>>>(d_spectrum, segment_pts_1x, segment_pts_1x, proc_cnt);
                CUFFTCHECK(hipfftExecC2R(planinv, (hipfftComplex *)d_spectrum, (hipfftReal *)d_sacdata));
                InvNormalize2DKernel<<<dimgrd_1x, dimblk_1x>>>(d_sacdata, segment_pts_1x, segment_pts_1x, proc_cnt, delta);
            }

            // Zero-Padding
            CUDACHECK(hipMemcpy2D(d_sacdata_2x, segment_pts_2x * sizeof(float), d_sacdata, segment_pts_1x * sizeof(float), segment_pts_1x * sizeof(float), proc_cnt, hipMemcpyDeviceToDevice));
            CUFFTCHECK(hipfftExecR2C(planfwd_2x, (hipfftReal *)d_sacdata_2x, (hipfftComplex *)d_spectrum_2x));
            FwdNormalize2DKernel<<<dimgrd_2x, dimblk_2x>>>(d_spectrum_2x, segment_pts_2x, segment_pts_2x, proc_cnt, delta);
            cisnan2DKernel<<<dimgrd_2x, dimblk_2x>>>(d_spectrum_2x, segment_pts_2x, segment_pts_2x, proc_cnt);

            // Copy data from d_spectrum back to h_spectrum
            CUDACHECK(hipMemcpy2D(h_spectrum + done_step * nspec_output, nstep_valid * nspec_output * sizeof(hipComplex),
                                   d_spectrum_2x, segment_pts_2x * sizeof(hipComplex),
                                   nspec_output * sizeof(complex), proc_cnt, hipMemcpyDeviceToHost));

            done_step++;
        }

        // End loop of each segment, Writing the output spectrum
        parallel_write_spec(write_pool, proc_cnt, pInOutList, argument.thread_num);
        destroy_threadpool_write(write_pool);
        destroy_threadpool_read(read_pool);
    }
    // Free memory
    hipfftDestroy(planfwd);
    hipfftDestroy(planinv);
    hipfftDestroy(planfwd_2x);
    GpuFree((void **)&d_sacdata);
    GpuFree((void **)&d_spectrum);
    GpuFree((void **)&d_sacdata_2x);
    GpuFree((void **)&d_spectrum_2x);
    GpuFree((void **)&d_filtered_sacdata);
    GpuFree((void **)&d_total_sacdata);
    GpuFree((void **)&d_responses);
    GpuFree((void **)&d_weight);
    GpuFree((void **)&d_tmp);
    GpuFree((void **)&d_tmp_weight);
    GpuFree((void **)&d_sum);
    GpuFree((void **)&d_isum);
    CpuFree((void **)&h_sacdata);
    CpuFree((void **)&h_spectrum);
    CpuFree((void **)&pInOutList);
}